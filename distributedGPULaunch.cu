#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include "utilsGPU.h"
#include <hip/hip_runtime.h>

__global__
void assignPoints(Point* points, Point* centroids, double* sums, int* nPoints, int* k, int* length) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < *length) {
        Point* p = &(points[index]);
        p->minDist = __DBL_MAX__;  // Reset before checking
        for (int i = 0; i < *k; ++i) {
            double dist = 0;
            Point* c = &(centroids[i]);
            for (int j = 0; j < 11; j++) {
                dist += (c->items[j] - p->items[j]) * (c->items[j] - p->items[j]);
            }

            if (dist < p->minDist) {
                p->minDist = dist;
                p->cluster = i;
            }
        }

        // Technically second for loop
        int cluster = p->cluster;
        for (int i = 0; i < ITEM_NUM; ++i) {
            double value = p->items[i];
            int sumIndex = i * K_CLUSTERS + cluster;
            atomicAdd(&sums[sumIndex], value);
        }
        atomicAdd(&nPoints[cluster], 1);
        p->minDist = __DBL_MAX__;
    }
}

extern "C++" void initCuda(double **d_sums, Point **d_points, Point **d_centroids, int **d_nPoints, int **d_k, int k, int **d_size, int localPointsCount, Point *localPoints) {
    // Declare all GPU memory structures
    hipMalloc((void **)d_sums, ITEM_NUM * K_CLUSTERS * sizeof(double));
    
    hipMalloc((void **)d_points, localPointsCount * sizeof(Point));
    
    hipMalloc((void **)d_centroids, K_CLUSTERS * sizeof(Point));
    
    hipMalloc((void **)d_nPoints, K_CLUSTERS * sizeof(int));
    
    hipMalloc((void **)d_k, sizeof(int));
    hipMalloc((void **)d_size, sizeof(int));
    
    hipMemcpy(*d_k, &k, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(*d_size, &localPointsCount, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(*d_points, localPoints, localPointsCount * sizeof(Point), hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA Error in initCuda: " << hipGetErrorString(err) << std::endl;
    }

}

extern "C++" void launchCuda(Point **d_centroids, Point *pointsCentroid, Point **d_points, double *sums, double **d_sums, int **d_nPoints, int *nPoints, int **d_k, int **d_size, int numBlocks, int blockSize) {
    hipMemcpy(*d_centroids, pointsCentroid, K_CLUSTERS * sizeof(Point), hipMemcpyHostToDevice);
    hipMemcpy(*d_sums, sums, ITEM_NUM * K_CLUSTERS * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(*d_nPoints, nPoints, K_CLUSTERS * sizeof(int), hipMemcpyHostToDevice);

    int checkSize = -1;
    hipMemcpy(&checkSize, *d_size, sizeof(int), hipMemcpyDeviceToHost);
    std::cout << "Size is: " << checkSize << std::endl;

    assignPoints<<<numBlocks, blockSize>>>(*d_points, *d_centroids, *d_sums, *d_nPoints, *d_k, *d_size);
    
    // Just finished kernel call. Check for errors, sync.
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA Error in launchCuda before function: " << hipGetErrorString(err) << std::endl;
    }
    
    // Copy needed memory back to host
    hipMemcpy(sums, *d_sums, ITEM_NUM * K_CLUSTERS * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(nPoints, *d_nPoints, K_CLUSTERS * sizeof(int), hipMemcpyDeviceToHost);
}

extern "C++" void getCudaPointsAndFree(Point *localPoints, Point **d_points, int localPointsCount, Point **d_centroids, int **d_k, int **d_size, double **d_sums, int **d_nPoints) {
    
    hipMemcpy(localPoints, *d_points, localPointsCount * sizeof(Point), hipMemcpyDeviceToHost);

    hipFree(*d_points);
    hipFree(*d_centroids);
    hipFree(*d_k);
    hipFree(*d_size);
    hipFree(*d_sums);
    hipFree(*d_nPoints);
}

extern "C++" void assignGPU(int rank) {
    hipSetDevice(rank);
}