#include "hip/hip_runtime.h"
#include <vector>
#include <string>
#include <iostream>
#include <numeric>
#include <algorithm>
#include <random>
#include <omp.h>
#include <stdexcept>
#include <hip/hip_runtime.h>
#include "utilsGPU.h"
#define K_CLUSTERS 5

__global__
void assignPoints(Point* points, Point* centroids, double* sums, int* nPoints, int* k, int* length) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < *length) {
        Point* p = &(points[index]);
        p->minDist = __DBL_MAX__;  // Reset before checking
        for (int i = 0; i < *k; ++i) {
            double dist = 0;
            Point* c = &(centroids[i]);
            for (int j = 0; j < 11; j++) {
                dist += (c->items[j] - p->items[j]) * (c->items[j] - p->items[j]);
            }

            if (dist < p->minDist) {
                p->minDist = dist;
                p->cluster = i;
            }
        }

        // Technically second for loop
        int cluster = p->cluster;
        for (int i = 0; i < ITEM_NUM; ++i) {
            double value = p->items[i];
            int sumIndex = i * K_CLUSTERS + cluster;
            atomicAdd(&sums[sumIndex], value);
        }
        atomicAdd(&nPoints[cluster], 1);
        p->minDist = __DBL_MAX__;
    }
}


void kMeans(Point* pointsArray, int epochs, int k, int thread_num) {

    // Initialize centroids
    Point pointsCentroid[K_CLUSTERS];

    std::vector<int> indices(DATA_NUM);
    std::iota(indices.begin(), indices.end(), 0);
    std::shuffle(indices.begin(), indices.end(), std::default_random_engine(100));
    
    for (int i = 0; i < k; ++i) {
        pointsCentroid[i] = Point(pointsArray[indices[i]].items);
    }

    // Declare arrays for points and centroids
    int size = DATA_NUM;

    double sums[ITEM_NUM][K_CLUSTERS] = {0.0};

    // Declare all GPU memory structures
    double *d_sums;
    hipMalloc((void **)&d_sums, ITEM_NUM * K_CLUSTERS * sizeof(double));
    
    Point *d_points;
    hipMalloc((void **)&d_points, DATA_NUM * sizeof(Point));
    
    Point *d_centroids;
    hipMalloc((void **)&d_centroids, K_CLUSTERS * sizeof(Point));
    
    int *d_nPoints;
    hipMalloc((void **)&d_nPoints, K_CLUSTERS * sizeof(int));
    
    int *d_k, *d_size;
    hipMalloc((void **)&d_k, sizeof(int));
    hipMalloc((void **)&d_size, sizeof(int));
    
    int blockSize = 16;
    int numBlocks = (size + blockSize - 1) / blockSize;
    
    hipMemcpy(d_k, &k, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_size, &size, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_points, pointsArray, DATA_NUM * sizeof(Point), hipMemcpyHostToDevice);
    
    // Run kmeans algorithm
    for (int x = 0; x < epochs; ++x) {

        int nPoints[K_CLUSTERS] = {0};
        
        // Fill sums array with zeroes (replaces second for loop)
        std::fill(&sums[0][0], &sums[0][0] + ITEM_NUM * K_CLUSTERS, 0.0);

        // Update GPU memory structures
        hipMemcpy(d_centroids, pointsCentroid, K_CLUSTERS * sizeof(Point), hipMemcpyHostToDevice);
        hipMemcpy(d_sums, &sums[0][0], ITEM_NUM * K_CLUSTERS * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_nPoints, nPoints, K_CLUSTERS * sizeof(int), hipMemcpyHostToDevice);

        assignPoints<<<numBlocks, blockSize>>>(d_points, d_centroids, d_sums, d_nPoints, d_k, d_size);

        // Just finished kernel call. Check for errors, sync.
        hipDeviceSynchronize();
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
        }
        
        // Copy needed memory back to host
        hipMemcpy(&sums[0][0], d_sums, ITEM_NUM * K_CLUSTERS * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(nPoints, d_nPoints, K_CLUSTERS * sizeof(int), hipMemcpyDeviceToHost);
                
        // Find mean of all points
        #pragma omp parallel for num_threads(thread_num)
        for (int i = 0; i < K_CLUSTERS; ++i) {
            if (nPoints[i] == 0) continue;
            for (int j = 0; j < ITEM_NUM; ++j) {
                pointsCentroid[i].items[j] = sums[j][i] / nPoints[i];
            }
            
        }
        
    }

    // Copy points back to host, copy array back into vector
    hipMemcpy(pointsArray, d_points, DATA_NUM * sizeof(Point), hipMemcpyDeviceToHost);

    // Free all memory
    hipFree(d_points);
    hipFree(d_centroids);
    hipFree(d_k);
    hipFree(d_size);
    hipFree(d_sums);
    hipFree(d_nPoints);
}


int main() {
    
    Point* points = readCSVNormalized("data/tracks_features.csv");
        
    // for (int i = 0; i < 3; i++) {
    //     std::cout << "Start" << std::endl;
    //     auto start2 = startTimerCPU();
    //     auto start = startTimerWall();
        
        
    //     endTimerWall(start);
    //     endTimerCPU(start2);
    // }

    kMeans(points, 5, K_CLUSTERS, 5);
    
    writeToCSV(points, "data/output_gpu.csv");

    compareFiles("data/output_normalized.csv", "data/output_gpu.csv");
    
    return 0;
}

// First execution is notably slower then others--this may be due to memory loading onto GPU?

// Time taken to run (wall clock): [1.587205, 1.824759, 1.607862] seconds (block size 512 threads)
// Time taken to run (cpu clock): [1.58146, 1.59184, 1.59481] seconds (block size 512 threads)

// Time taken to run (wall clock): [1.613788, 1.631238, 1.620053] seconds (block size 256 threads)
// Time taken to run (cpu clock): [1.60671, 1.62136, 1.60614] seconds (block size 256 threads)

// Time taken to run (wall clock): [1.603406, 1.630459, 1.605275] seconds (block size 128 threads)
// Time taken to run (cpu clock): [1.59254, 1.62097, 1.59973] seconds (block size 128 threads)

// Time taken to run (wall clock): [1.607722, 1.459259, 1.469630] seconds (block size 64 threads)
// Time taken to run (cpu clock): [1.56956, 1.45311, 1.46486] seconds (block size 64 threads)

// Time taken to run (wall clock): [0.536088, 0.530974, 0.531105] seconds (block size 32 threads)
// Time taken to run (cpu clock): [0.532402, 0.527215, 0.526539] seconds (block size 32 threads)

// Time taken to run (wall clock): [0.719224, 0.721331, 0.711376] seconds (block size 16 threads)
// Time taken to run (cpu clock): [0.715885, 0.71655, 0.709268] seconds (block size 16 threads)