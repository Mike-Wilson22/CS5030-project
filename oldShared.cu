#include "hip/hip_runtime.h"
#include <vector>
#include <string>
#include <fstream>
#include <iostream>
#include <sstream>
#include <numeric>
#include <algorithm>
#include <random>
// #include <omp.h>
#include <hip/hip_runtime.h>
#include "utilsGPU.h"
#define K_CLUSTERS 5

__global__
void assignPoints(Point* points, Point* centroids, int* k, int* length) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < *length) {
        Point* p = &(points[index]);
        p->minDist = __DBL_MAX__;  // Reset before checking
        for (int i = 0; i < *k; ++i) {
            double dist = 0;
            Point* c = &(centroids[i]);
            for (int j = 0; j < 11; j++) {
                dist += (c->items[j] - p->items[j]) * (c->items[j] - p->items[j]);
            }
            // double dist = centroids[i].distance(p);
            if (dist < p->minDist) {
                p->minDist = dist;
                p->cluster = i;
            }
        }
    }
}

void kMeans(std::vector<Point>* points, int epochs, int k, int thread_num) {

    // Initialize centroids
    std::vector<Point> centroids;
    std::vector<Point> centroids2;
    std::vector<int> indices(points->size());
    std::iota(indices.begin(), indices.end(), 0);
    std::shuffle(indices.begin(), indices.end(), std::default_random_engine(100));
    
    for (int i = 0; i < k; ++i) {
        centroids.push_back(Point(points->at(indices[i]).items));
    }

    for (int i = 0; i < k; ++i) {
        centroids2.push_back(Point(points->at(indices[i]).items));
    }
    
    int size = DATA_NUM;
    Point pointsArray[DATA_NUM];
    std::cout << "size: " << points->size() << std::endl;
    std::copy(points->begin(), points->end(), pointsArray);
    Point* pointsPointer = pointsArray;

    Point pointsCentroid[5];
    std::copy(centroids.begin(), centroids.end(), pointsCentroid);
    Point* centroidPointer = pointsCentroid;
    
    
    Point *d_points;
    hipMalloc((void **)&d_points, DATA_NUM * sizeof(Point));

    Point *d_centroids;
    hipMalloc((void **)&d_centroids, centroids.size() * sizeof(Point));

    int *d_k, *d_size;
    hipMalloc((void **)&d_k, sizeof(int));
    hipMalloc((void **)&d_size, sizeof(int));
    

    hipMemcpy(d_k, &k, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_size, &size, sizeof(int), hipMemcpyHostToDevice);
    // Run kmeans algorithm
    for (int x = 0; x < epochs; ++x) {

        // Assign all points to initial clusters
        // #pragma omp parallel for num_threads(thread_num)
        // for (int j = 0; j < points->size(); ++j) {
        //     Point* p = &(points->at(j));
        //     p->minDist = __DBL_MAX__;  // Reset before checking
        //     for (int i = 0; i < k; ++i) {
        //         double dist = centroids.at(i).distance(p);
        //         if (dist < p->minDist) {
        //             p->minDist = dist;
        //             p->cluster = i;
        //         }
        //     }
        // }

        // Setup data for GPU
        std::cout << "copying memory " << std::endl;
        std::copy(centroids.begin(), centroids.end(), pointsCentroid);
        hipMemcpy(d_centroids, centroidPointer, centroids.size() * sizeof(Point), hipMemcpyHostToDevice);
        hipMemcpy(d_points, pointsPointer, DATA_NUM * sizeof(Point), hipMemcpyHostToDevice);

        std::cout << "calling kernel function " << std::endl;
        // assignPoints<<<ceil(size/256), 256>>>(d_points, d_centroids, d_k, d_size);
        int blockSize = 256;
        int numBlocks = (size + blockSize - 1) / blockSize;
        assignPoints<<<numBlocks, blockSize>>>(d_points, d_centroids, d_k, d_size);

        //Just finished kernel call. Check for errors, sync.
        hipDeviceSynchronize();
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
        }

        std::cout << "copying memory back " << std::endl;
        hipMemcpy(pointsPointer, d_points, DATA_NUM * sizeof(Point), hipMemcpyDeviceToHost);

        
        std::cout << "run omp stuff " << std::endl;

        for (int j = 0; j < points->size(); ++j) {
            Point* p = &(points->at(j));
            p->minDist = __DBL_MAX__;  // Reset before checking
            for (int i = 0; i < k; ++i) {
                double dist = centroids2.at(i).distance(p);
                if (dist < p->minDist) {
                    p->minDist = dist;
                    p->cluster = i;
                }
            }
        }

        for (int i = 0; i < 10; i++) {
            Point point1 = pointsArray[i];
            Point point2 = points->at(i);

            if (point1.minDist != point2.minDist) {
                std::cout << "minDist1: " << point1.minDist << ", minDist2: " << point2.minDist << std::endl;
            }
            if (point1.cluster != point2.cluster) {
                std::cout << "cluster1: " << point1.cluster << ", cluster2: " << point2.cluster << std::endl;
            }
        }

        // Initialize vectors to help with calculating means
        std::vector<std::vector<double>> sums;
        std::vector<std::vector<double>> sums2;
        
        for (int j = 0; j < ITEM_NUM; ++j) {
            std::vector<double> sum;
            for (int x = 0; x < k; ++x) {
                sum.push_back(0.0);
            }
            sums.push_back(sum);
        }

        for (int j = 0; j < ITEM_NUM; ++j) {
            std::vector<double> sum;
            for (int x = 0; x < k; ++x) {
                sum.push_back(0.0);
            }
            sums2.push_back(sum);
        }
        
        // #pragma omp parallel for num_threads(thread_num)
        for (int i = 0; i < sums2.size(); ++i) {
            for (int j = 0; j < size; ++j) {
                sums2[i][points->at(j).cluster] += points->at(j).items[i];
            }
        }

        for (int i = 0; i < sums.size(); ++i) {
            for (int j = 0; j < size; ++j) {
                sums[i][pointsArray[j].cluster] += pointsArray[j].items[i];
            }
        }
        
        int nPoints[K_CLUSTERS] = {0};
        int nPoints2[K_CLUSTERS] = {0};
        // #pragma omp parallel num_threads(thread_num)
        {
            // int nPointsOMP[k] = {0};
            // #pragma omp for
            for (int i = 0; i < points->size(); ++i) {
                nPoints[pointsArray[i].cluster]++;
                pointsArray[i].minDist = __DBL_MAX__;
            }

            for (int i = 0; i < points->size(); ++i) {
                nPoints2[points->at(i).cluster]++;
                points->at(i).minDist = __DBL_MAX__;
            }
            
            // #pragma omp critical
            // {
            //     for (int i = 0; i < k; ++i) {
            //         nPoints[i] += nPointsOMP[i];
            //     }
            // }
        }
        
        // Find mean of all points
        // #pragma omp parallel for num_threads(thread_num)
        for (int i = 0; i < centroids.size(); ++i) {
            if (nPoints[i] == 0) continue;
            for (int j = 0; j < sums.size(); ++j) {
                centroids.at(i).items[j] = sums[j][i] / nPoints[i];
            }
            
        }

        for (int i = 0; i < centroids2.size(); ++i) {
            if (nPoints2[i] == 0) continue;
            for (int j = 0; j < sums2.size(); ++j) {
                centroids2.at(i).items[j] = sums2[j][i] / nPoints2[i];
            }
            
        }
    }
    for (int i = 0; i < DATA_NUM; i++) {
        points->at(i) = pointsArray[i];
    }
    hipFree(d_points);
    hipFree(d_centroids);
    hipFree(d_k);
    hipFree(d_size);
}


int main() {
    
    std::vector<Point> points = readCSV("data/tracks_features.csv");
    if (points.empty()) {
        std::cerr << "Error: No points loaded from CSV." << std::endl;
        return 1;
    }    
    
    kMeans(&points, 5, 5, 5);
    std::cout << "finished kmeans " << std::endl;
    writeToCSV(&points, "data/output_gpu.csv");

    compareFiles("data/output.csv", "data/output_gpu.csv");
    
    return 0;
}